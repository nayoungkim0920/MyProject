#include "hip/hip_runtime.h"
//imageProecssing.cu
#include "imageProcessing.cuh"

__device__ void rotatePixel(int x, int y, int cols, int rows, int channels, const unsigned char* input, unsigned char* output) {
    if (x < cols && y < rows) {
        for (int c = 0; c < channels; ++c) {
            output[(x * rows + (rows - 1 - y)) * channels + c] = input[(y * cols + x) * channels + c];
        }
    }
}

__global__ void rotateImageKernel(const unsigned char* input, unsigned char* output, int cols, int rows, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        rotatePixel(x, y, cols, rows, channels, input, output);
    }
}

__global__ void resizeImageKernel(const unsigned char* input, unsigned char* output, int oldWidth, int oldHeight, int newWidth, int newHeight, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < newWidth && y < newHeight) {
        float x_ratio = oldWidth / (float)newWidth;
        float y_ratio = oldHeight / (float)newHeight;
        int px = floor(x * x_ratio);
        int py = floor(y * y_ratio);

        for (int c = 0; c < channels; ++c) {
            output[(y * newWidth + x) * channels + c] = input[(py * oldWidth + px) * channels + c];
        }
    }
}

__global__ void grayScaleImageKernel(const unsigned char* input, unsigned char* output, int cols, int rows) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int idx = y * cols + x;
        unsigned char b = input[idx * 3 + 0];
        unsigned char g = input[idx * 3 + 1];
        unsigned char r = input[idx * 3 + 2];
        output[idx] = 0.299f * r + 0.587f * g + 0.114f * b;  // �׷��̽����� ��ȯ
    }
}

__global__ void cannyEdgesKernel(const unsigned char* input, unsigned char* output, int cols, int rows) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int idx = y * cols + x;
        float gradientX = 0.0f, gradientY = 0.0f;

        // Calculate gradients (Sobel operators)
        if (x > 0 && x < cols - 1 && y > 0 && y < rows - 1) {
            gradientX = -1.0f * input[(y - 1) * cols + (x - 1)] + 1.0f * input[(y - 1) * cols + (x + 1)]
                - 2.0f * input[y * cols + (x - 1)] + 2.0f * input[y * cols + (x + 1)]
                - 1.0f * input[(y + 1) * cols + (x - 1)] + 1.0f * input[(y + 1) * cols + (x + 1)];

            gradientY = -1.0f * input[(y - 1) * cols + (x - 1)] - 2.0f * input[(y - 1) * cols + x] - 1.0f * input[(y - 1) * cols + (x + 1)]
                + 1.0f * input[(y + 1) * cols + (x - 1)] + 2.0f * input[(y + 1) * cols + x] + 1.0f * input[(y + 1) * cols + (x + 1)];
        }

        // Calculate gradient magnitude
        float gradientMagnitude = sqrtf(gradientX * gradientX + gradientY * gradientY);

        // Apply hysteresis thresholding to detect edges
        if (gradientMagnitude > 50) {  // Adjust this threshold as needed
            output[idx] = 255;
        }
        else {
            output[idx] = 0;
        }
    }
}

__global__ void gaussianBlurKernel(const unsigned char* input, unsigned char* output, int cols, int rows, int kernelSize, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int halfSize = kernelSize / 2;
        float sum = 0.0f;

        // Apply Gaussian blur using the kernel size
        for (int c = 0; c < channels; ++c) {
            sum = 0.0f;

            for (int i = -halfSize; i <= halfSize; ++i) {
                for (int j = -halfSize; j <= halfSize; ++j) {
                    int offsetX = x + i;
                    int offsetY = y + j;

                    if (offsetX >= 0 && offsetX < cols && offsetY >= 0 && offsetY < rows) {
                        float weight = exp(-(i * i + j * j) / (2.0f * kernelSize * kernelSize));
                        sum += weight * input[(offsetY * cols + offsetX) * channels + c];
                    }
                }
            }

            output[(y * cols + x) * channels + c] = static_cast<unsigned char>(sum);
        }
    }
}

__device__ void device_sort(unsigned char* values, int length) {
    // Sorting implementation using device-specific method
    // Example: bubble sort
    for (int i = 0; i < length - 1; ++i) {
        for (int j = 0; j < length - i - 1; ++j) {
            if (values[j] > values[j + 1]) {
                unsigned char temp = values[j];
                values[j] = values[j + 1];
                values[j + 1] = temp;
            }
        }
    }
}

__global__ void medianFilterKernel(const unsigned char* input, unsigned char* output, int cols, int rows, int channels, int kernelSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int halfSize = kernelSize / 2;
        int kernelLength = kernelSize * kernelSize;

        // ���� ������ �ȼ��� �ε��� ���
        int idx = (y * cols + x) * channels;

        // �߰����� ã�� ���� �ӽ� ���� ����
        unsigned char values[25]; // �ִ� kernelSize�� 5�� ����

        // �� ä�ο� ���� median ���� ����
        for (int c = 0; c < channels; ++c) {
            // ���� �ʱ�ȭ
            for (int i = 0; i < kernelLength; ++i) {
                int offsetX = x + (i % kernelSize) - halfSize;
                int offsetY = y + (i / kernelSize) - halfSize;

                // ��� ó��
                offsetX = max(0, min(cols - 1, offsetX));
                offsetY = max(0, min(rows - 1, offsetY));

                values[i] = input[(offsetY * cols + offsetX) * channels + c];
            }

            // ���� ���� �� �߰��� ��� (���� ���� �˰��� ȣ��)
            device_sort(values, kernelLength);

            output[idx + c] = values[kernelLength / 2];
        }
    }
}


extern "C" void callRotateImageCUDA(cv::Mat & inputImage) {
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t imageSize = cols * rows * channels * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, imageSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, imageSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, imageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    //host�� kernel���и��Ͽ� imageProcessing.cpp�θ�������� �Ʒ��� ���������ʾ� �ٽ� .cu���Ͽ� ������
    rotateImageKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows, channels);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    err = hipMemcpy(inputImage.data, d_outputImage, imageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

extern "C" void callResizeImageCUDA(cv::Mat & inputImage, int newWidth, int newHeight) {
    int oldWidth = inputImage.cols;
    int oldHeight = inputImage.rows;
    int channels = inputImage.channels();

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t oldImageSize = oldWidth * oldHeight * channels * sizeof(uchar);
    size_t newImageSize = newWidth * newHeight * channels * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, oldImageSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, newImageSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, oldImageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((newWidth + threadsPerBlock.x - 1) / threadsPerBlock.x, (newHeight + threadsPerBlock.y - 1) / threadsPerBlock.y);

    resizeImageKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, oldWidth, oldHeight, newWidth, newHeight, channels);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    cv::Mat outputImage(newHeight, newWidth, inputImage.type());
    err = hipMemcpy(outputImage.data, d_outputImage, newImageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
    }
    else {
        inputImage = outputImage;
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

extern "C" void callGrayScaleImageCUDA(cv::Mat & inputImage) {
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    if (channels != 3) {
        std::cerr << "Input image must be a 3-channel BGR image." << std::endl;
        return;
    }

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t inputSize = cols * rows * channels * sizeof(uchar);
    size_t outputSize = cols * rows * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, inputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, inputSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    grayScaleImageKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    cv::Mat outputImage(rows, cols, CV_8UC1);
    err = hipMemcpy(outputImage.data, d_outputImage, outputSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
    }
    else {
        inputImage = outputImage;
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

void callCannyEdgesCUDA(cv::Mat& inputImage) {
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    if (channels != 1) {
        std::cerr << "Input image must be a single-channel grayscale image." << std::endl;
        return;
    }

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t inputSize = cols * rows * sizeof(uchar);
    size_t outputSize = cols * rows * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, inputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, inputSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Initialize output image to 0 (optional, for safety)
    err = hipMemset(d_outputImage, 0, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA memset error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    // Launch CUDA kernel
    cannyEdgesKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    // Copy result back to host
    err = hipMemcpy(inputImage.data, d_outputImage, outputSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

extern "C" void callGaussianBlur(cv::Mat & inputImage, int kernelSize) {
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t inputSize = cols * rows * channels * sizeof(uchar);
    size_t outputSize = cols * rows * channels * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, inputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, inputSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    gaussianBlurKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows, kernelSize, channels);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    err = hipMemcpy(inputImage.data, d_outputImage, outputSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

extern "C" void callMedianFilterCUDA(cv::Mat & inputImage)
{
    // �̹����� �ʺ�, ����, ä�� �� Ȯ��
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    // GPU �޸� �Ҵ�
    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t inputSize = cols * rows * channels * sizeof(uchar);
    size_t outputSize = cols * rows * channels * sizeof(uchar);

    hipError_t err;

    err = hipMalloc(&d_inputImage, inputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    // CPU���� GPU�� �̹��� ������ ����
    err = hipMemcpy(d_inputImage, inputImage.data, inputSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy H2D error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    // CUDA ������ ���� ����
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // ä�� ���� ���� ������ Ŀ�� ����
    if (channels == 1 || channels == 3) {
        medianFilterKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows, channels, 5);
    }
    else {
        std::cerr << "Unsupported number of channels: " << channels << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    // CUDA Ŀ�� ���� ���� Ȯ��
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    // GPU���� ó�� �Ϸ� ���
    hipDeviceSynchronize();

    // GPU���� CPU�� ��� �̹��� ����
    cv::Mat outputImage(rows, cols, inputImage.type());
    err = hipMemcpy(outputImage.data, d_outputImage, outputSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy D2H error: " << hipGetErrorString(err) << std::endl;
    }
    else {
        inputImage = outputImage.clone();
    }

    // �޸� ����
    hipFree(d_inputImage);
    hipFree(d_outputImage);
}