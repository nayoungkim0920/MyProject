#include "hip/hip_runtime.h"
//imageProecssing.cu
#include "imageProcessing.cuh"

#define CUDA_CHECK_ERROR(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
        return; \
    }

__device__ void rotatePixel(int x, int y, int cols, int rows, int channels, const unsigned char* input, unsigned char* output) {
    if (x < cols && y < rows) {
        for (int c = 0; c < channels; ++c) {
            output[(x * rows + (rows - 1 - y)) * channels + c] = input[(y * cols + x) * channels + c];
        }
    }
}

__device__ float gaussian(float x, float sigma) {
    return expf(-(x * x) / (2 * sigma * sigma));
}

__global__ void resizeImageKernel(const unsigned char* input, unsigned char* output, int oldWidth, int oldHeight, int newWidth, int newHeight, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < newWidth && y < newHeight) {
        float x_ratio = oldWidth / (float)newWidth;
        float y_ratio = oldHeight / (float)newHeight;
        int px = floor(x * x_ratio);
        int py = floor(y * y_ratio);

        for (int c = 0; c < channels; ++c) {
            output[(y * newWidth + x) * channels + c] = input[(py * oldWidth + px) * channels + c];
        }
    }
}

__global__ void grayScaleImageKernel(const unsigned char* input, unsigned char* output, int cols, int rows) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int idx = y * cols + x;
        unsigned char b = input[idx * 3 + 0];
        unsigned char g = input[idx * 3 + 1];
        unsigned char r = input[idx * 3 + 2];
        output[idx] = 0.299f * r + 0.587f * g + 0.114f * b;  // �׷��̽����� ��ȯ
    }
}

__global__ void cannyEdgesKernel(const unsigned char* input, unsigned char* output, int cols, int rows, int channels, bool isColor) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int idx = y * cols + x;
        float gradientX = 0.0f, gradientY = 0.0f;

        // Calculate gradients (Sobel operators)
        if (x > 0 && x < cols - 1 && y > 0 && y < rows - 1) {
            for (int c = 0; c < (isColor ? 3 : 1); c++) {
                int offset = c * rows * cols;
                gradientX += -1.0f * input[offset + (y - 1) * cols + (x - 1)] + 1.0f * input[offset + (y - 1) * cols + (x + 1)]
                    - 2.0f * input[offset + y * cols + (x - 1)] + 2.0f * input[offset + y * cols + (x + 1)]
                    - 1.0f * input[offset + (y + 1) * cols + (x - 1)] + 1.0f * input[offset + (y + 1) * cols + (x + 1)];

                gradientY += -1.0f * input[offset + (y - 1) * cols + (x - 1)] - 2.0f * input[offset + (y - 1) * cols + x] - 1.0f * input[offset + (y - 1) * cols + (x + 1)]
                    + 1.0f * input[offset + (y + 1) * cols + (x - 1)] + 2.0f * input[offset + (y + 1) * cols + x] + 1.0f * input[offset + (y + 1) * cols + (x + 1)];
            }
        }

        // Calculate gradient magnitude
        float gradientMagnitude = sqrtf(gradientX * gradientX + gradientY * gradientY);

        // Apply hysteresis thresholding to detect edges
        if (gradientMagnitude > 50) {  // Adjust this threshold as needed
            if (isColor) {
                output[idx * 3] = 0;       // Blue
                output[idx * 3 + 1] = 255; // Green
                output[idx * 3 + 2] = 0;   // Red
            }
            else {
                output[idx] = 255;
            }
        }
        else {
            if (isColor) {
                output[idx * 3] = input[idx * 3];
                output[idx * 3 + 1] = input[idx * 3 + 1];
                output[idx * 3 + 2] = input[idx * 3 + 2];
            }
            else {
                output[idx] = 0;
            }
        }
    }
}

__global__ void gaussianBlurKernel(const unsigned char* input, unsigned char* output, int cols, int rows, int kernelSize, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int halfSize = kernelSize / 2;
        float sum = 0.0f;
        float normalization = 0.0f;

        for (int c = 0; c < channels; ++c) {
            sum = 0.0f;
            normalization = 0.0f;

            for (int i = -halfSize; i <= halfSize; ++i) {
                for (int j = -halfSize; j <= halfSize; ++j) {
                    int offsetX = x + i;
                    int offsetY = y + j;

                    if (offsetX >= 0 && offsetX < cols && offsetY >= 0 && offsetY < rows) {
                        float weight = expf(-(i * i + j * j) / (2.0f * halfSize * halfSize));
                        sum += weight * input[(offsetY * cols + offsetX) * channels + c];
                        normalization += weight;
                    }
                }
            }

            output[(y * cols + x) * channels + c] = static_cast<unsigned char>(sum / normalization);
        }
    }
}

__device__ void device_sort(unsigned char* values, int length) {
    // Sorting implementation using device-specific method
    // Example: bubble sort
    for (int i = 0; i < length - 1; ++i) {
        for (int j = 0; j < length - i - 1; ++j) {
            if (values[j] > values[j + 1]) {
                unsigned char temp = values[j];
                values[j] = values[j + 1];
                values[j + 1] = temp;
            }
        }
    }
}

__global__ void medianFilterKernel(const unsigned char* input, unsigned char* output, int cols, int rows, int channels, int kernelSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int halfSize = kernelSize / 2;
        int kernelLength = kernelSize * kernelSize;

        // ���� ������ �ȼ��� �ε��� ���
        int idx = (y * cols + x) * channels;

        // �߰����� ã�� ���� �ӽ� ���� ����
        unsigned char values[25]; // �ִ� kernelSize�� 5�� ����

        // �� ä�ο� ���� median ���� ����
        for (int c = 0; c < channels; ++c) {
            // ���� �ʱ�ȭ
            for (int i = 0; i < kernelLength; ++i) {
                int offsetX = x + (i % kernelSize) - halfSize;
                int offsetY = y + (i / kernelSize) - halfSize;

                // ��� ó��
                offsetX = max(0, min(cols - 1, offsetX));
                offsetY = max(0, min(rows - 1, offsetY));

                values[i] = input[(offsetY * cols + offsetX) * channels + c];
            }

            // ���� ���� �� �߰��� ��� (���� ���� �˰��� ȣ��)
            device_sort(values, kernelLength);

            output[idx + c] = values[kernelLength / 2];
        }
    }
}

__global__ void laplacianFilterKernel(const unsigned char* input, unsigned char* output,
    int cols, int rows, size_t pitch, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 1 && x < cols - 1 && y >= 1 && y < rows - 1) {
        // Laplacian ���� ���
        int kernel[3][3] = {
            {0, -1, 0},
            {-1, 4, -1},
            {0, -1, 0}
        };

        for (int c = 0; c < channels; ++c) {
            float sum = 0.0f;

            // Ŀ�� ����
            for (int ky = -1; ky <= 1; ++ky) {
                for (int kx = -1; kx <= 1; ++kx) {
                    int pixelX = x + kx;
                    int pixelY = y + ky;
                    float pixelValue = input[pixelY * pitch + pixelX * channels + c];
                    sum += pixelValue * kernel[ky + 1][kx + 1];
                }
            }

            // ��� ����
            output[y * pitch + x * channels + c] = static_cast<unsigned char>(min(max(sum, 0.0f), 255.0f));
        }
    }
}
__global__ void bilateralKernel(
    const unsigned char* d_input,
    unsigned char* d_output,
    int width,
    int height,
    int pitch, // pitch �߰�
    int kernelSize,
    int channels,
    float sigmaColor,
    float sigmaSpace
) {
    // ���� �ȼ� ��ǥ
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int radius = kernelSize / 2;
    float colorSum[3] = { 0.0f, 0.0f, 0.0f }; // �� ä���� ���� �ջ��� ���� �迭
    float weightSum = 0.0f;

    // ���� �ȼ� �� ��������
    unsigned char inputPixel[3];
    for (int c = 0; c < channels; ++c) {
        inputPixel[c] = d_input[y * pitch + x * channels + c]; // pitch ���
    }

    // Ŀ�� ��ȸ
    for (int dy = -radius; dy <= radius; ++dy) {
        for (int dx = -radius; dx <= radius; ++dx) {
            int nx = min(max(x + dx, 0), width - 1);
            int ny = min(max(y + dy, 0), height - 1);

            unsigned char neighborPixel[3];
            for (int c = 0; c < channels; ++c) {
                neighborPixel[c] = d_input[ny * pitch + nx * channels + c]; // pitch ���
            }

            // ���� ���� �� ����ġ ���
            float colorDiff = 0.0f;
            for (int c = 0; c < channels; ++c) {
                float diff = inputPixel[c] - neighborPixel[c];
                colorDiff += diff * diff;
            }
            float colorWeight = expf(-colorDiff / (2.0f * sigmaColor * sigmaColor));

            float spatialDist = dx * dx + dy * dy;
            float spatialWeight = expf(-spatialDist / (2.0f * sigmaSpace * sigmaSpace));

            float weight = colorWeight * spatialWeight;
            weightSum += weight;

            for (int c = 0; c < channels; ++c) {
                colorSum[c] += neighborPixel[c] * weight;
            }
        }
    }

    // ��� �ȼ� �� ����
    for (int c = 0; c < channels; ++c) {
        d_output[y * pitch + x * channels + c] = static_cast<unsigned char>(min(max(colorSum[c] / weightSum, 0.0f), 255.0f));
    }
}

__global__ void sobelFilterKernel(const unsigned char* d_input, unsigned char* d_output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int idx = y * width + x;
    int sobelX[3][3] = { { -1, 0, 1 },
                         { -2, 0, 2 },
                         { -1, 0, 1 } };

    int sobelY[3][3] = { { -1, -2, -1 },
                         { 0, 0, 0 },
                         { 1, 2, 1 } };

    float gradientX = 0;
    float gradientY = 0;

    for (int ky = -1; ky <= 1; ++ky) {
        for (int kx = -1; kx <= 1; ++kx) {
            int px = min(max(x + kx, 0), width - 1);
            int py = min(max(y + ky, 0), height - 1);
            int pixelIdx = py * width + px;

            for (int c = 0; c < channels; ++c) {
                float pixelValue = d_input[pixelIdx * channels + c];
                gradientX += pixelValue * sobelX[ky + 1][kx + 1];
                gradientY += pixelValue * sobelY[ky + 1][kx + 1];
            }
        }
    }

    float magnitude = sqrtf(gradientX * gradientX + gradientY * gradientY);
    magnitude = min(max(magnitude, 0.0f), 255.0f); // Clip the values to be within 0-255

    if (channels == 1) {
        d_output[idx] = static_cast<unsigned char>(magnitude);
    }
    else if (channels == 3) {
        d_output[idx * 3 + 0] = static_cast<unsigned char>(magnitude); // Apply the same magnitude to all channels
        d_output[idx * 3 + 1] = static_cast<unsigned char>(magnitude);
        d_output[idx * 3 + 2] = static_cast<unsigned char>(magnitude);
    }
}

__global__ void rotateImageKernelR(const unsigned char* input, unsigned char* output, int cols, int rows, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        for (int c = 0; c < channels; ++c) {
            // ���������� 90�� ȸ��
            output[(x * rows + (rows - y - 1)) * channels + c] = input[(y * cols + x) * channels + c];
        }
    }
}

void callRotateImageCUDA_R(cv::Mat& inputImage, cv::Mat& outputImage) {
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t imageSize = cols * rows * channels * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, imageSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, imageSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, imageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    rotateImageKernelR << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows, channels);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    // ȸ���� �̹����� ũ�⸦ ����
    outputImage.create(cols, rows, inputImage.type());

    err = hipMemcpy(outputImage.data, d_outputImage, imageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

__global__ void rotateImageKernelL(const unsigned char* input, unsigned char* output, int cols, int rows, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        for (int c = 0; c < channels; ++c) {
            // �������� 90�� ȸ��
            output[((cols - 1 - x) * rows + y) * channels + c] = input[(y * cols + x) * channels + c];
        }
    }
}

void callRotateImageCUDA_L(cv::Mat& inputImage, cv::Mat& outputImage) {
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t inputSize = cols * rows * channels * sizeof(uchar);
    size_t outputSize = rows * cols * channels * sizeof(uchar); // ȸ�� �� �̹��� ũ��

    hipError_t err;
    err = hipMalloc(&d_inputImage, inputSize);
    CUDA_CHECK_ERROR(err);

    err = hipMalloc(&d_outputImage, outputSize);
    CUDA_CHECK_ERROR(err);

    err = hipMemcpy(d_inputImage, inputImage.data, inputSize, hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR(err);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    rotateImageKernelL << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows, channels);
    err = hipGetLastError();
    CUDA_CHECK_ERROR(err);

    hipDeviceSynchronize();

    // ȸ���� �̹����� ũ�⸦ ���� (�ʺ�� ���̰� �ٲ�)
    outputImage.create(cols, rows, inputImage.type());

    err = hipMemcpy(outputImage.data, d_outputImage, outputSize, hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR(err);

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

void callZoomImageCUDA(cv::Mat& inputImage, cv::Mat& outputImage, int newWidth, int newHeight) {
    int oldWidth = inputImage.cols;
    int oldHeight = inputImage.rows;
    int channels = inputImage.channels();

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t oldImageSize = oldWidth * oldHeight * channels * sizeof(uchar);
    size_t newImageSize = newWidth * newHeight * channels * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, oldImageSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, newImageSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, oldImageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((newWidth + threadsPerBlock.x - 1) / threadsPerBlock.x, (newHeight + threadsPerBlock.y - 1) / threadsPerBlock.y);

    resizeImageKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, oldWidth, oldHeight, newWidth, newHeight, channels);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    outputImage.create(newHeight, newWidth, inputImage.type());

    err = hipMemcpy(outputImage.data, d_outputImage, newImageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

void callGrayScaleImageCUDA(cv::Mat& inputImage, cv::Mat& outputImage) {
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    if (channels != 3) {
        std::cerr << "Input image must be a 3-channel BGR image." << std::endl;
        return;
    }

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t inputSize = cols * rows * channels * sizeof(uchar);
    size_t outputSize = cols * rows * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, inputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, inputSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    grayScaleImageKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    outputImage.create(rows, cols, CV_8UC1);
    err = hipMemcpy(outputImage.data, d_outputImage, outputSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

void callCannyEdgesCUDA(cv::Mat& inputImage, cv::Mat& outputImage) {
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();
    bool isColor = (channels == 3);

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t inputSize = cols * rows * channels * sizeof(uchar);
    size_t outputSize = cols * rows * (isColor ? channels : 1) * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, inputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, inputSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    cannyEdgesKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows, channels, isColor);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    if (isColor) {
        outputImage.create(rows, cols, CV_8UC3);
    }
    else {
        outputImage.create(rows, cols, CV_8UC1);
    }

    err = hipMemcpy(outputImage.data, d_outputImage, outputSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

void callGaussianBlurCUDA(cv::Mat& inputImage, cv::Mat& outputImage, int kernelSize) {
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t inputSize = cols * rows * channels * sizeof(uchar);
    size_t outputSize = cols * rows * channels * sizeof(uchar);

    hipError_t err;

    // CUDA �޸� �Ҵ�
    err = hipMalloc(&d_inputImage, inputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error for inputImage: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error for outputImage: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    // CUDA �޸𸮷� ������ ����
    err = hipMemcpy(d_inputImage, inputImage.data, inputSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error for inputImage: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    // CUDA Ŀ�� ȣ�� ����
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Gaussian blur Ŀ�� ȣ��
    gaussianBlurKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows, kernelSize, channels);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    // ��� �̹��� ���� �� ������ ����
    outputImage.create(rows, cols, inputImage.type());
    err = hipMemcpy(outputImage.data, d_outputImage, outputSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error for outputImage: " << hipGetErrorString(err) << std::endl;
    }

    // �޸� ����
    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

void callMedianFilterCUDA(cv::Mat & inputImage, cv::Mat& outputImage)
{
    // �̹����� �ʺ�, ����, ä�� �� Ȯ��
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    // GPU �޸� �Ҵ�
    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t inputSize = cols * rows * channels * sizeof(uchar);
    size_t outputSize = cols * rows * channels * sizeof(uchar);

    hipError_t err;

    err = hipMalloc(&d_inputImage, inputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    // CPU���� GPU�� �̹��� ������ ����
    err = hipMemcpy(d_inputImage, inputImage.data, inputSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy H2D error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    // CUDA ������ ���� ����
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // ä�� ���� ���� ������ Ŀ�� ����
    if (channels == 1 || channels == 3) {
        medianFilterKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows, channels, 5);
    }
    else {
        std::cerr << "Unsupported number of channels: " << channels << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    // CUDA Ŀ�� ���� ���� Ȯ��
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    // GPU���� ó�� �Ϸ� ���
    hipDeviceSynchronize();

    // GPU���� CPU�� ��� �̹��� ����
    cv::Mat gpuOutputImage(rows, cols, inputImage.type());
    err = hipMemcpy(gpuOutputImage.data, d_outputImage, outputSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy D2H error: " << hipGetErrorString(err) << std::endl;
    }
    //else {
    //    inputImage = outputImage.clone();
    //}
    outputImage = gpuOutputImage.clone();

    // �޸� ����
    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

void callLaplacianFilterCUDA(cv::Mat& inputImage, cv::Mat& outputImage) {
    int width = inputImage.cols;
    int height = inputImage.rows;
    int channels = inputImage.channels();

    unsigned char* d_input;
    unsigned char* d_output;
    size_t pitch;

    // CUDA �޸� �Ҵ�
    hipMallocPitch(&d_input, &pitch, width * channels * sizeof(unsigned char), height);
    hipMallocPitch(&d_output, &pitch, width * channels * sizeof(unsigned char), height);

    // �Է� �̹��� ����
    hipMemcpy2D(d_input, pitch, inputImage.ptr(), inputImage.step[0], width * channels * sizeof(unsigned char), height, hipMemcpyHostToDevice);

    // CUDA Ŀ�� ���� ����
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // CUDA Ŀ�� ȣ��
    laplacianFilterKernel << <gridSize, blockSize >> > (d_input, d_output, width, height, pitch, channels);

    // CUDA ���� üũ
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        return;
    }

    // ��� �̹��� ����
    outputImage.create(height, width, inputImage.type()); // �ùٸ� ���̿� �ʺ�� �̹��� ����
    hipMemcpy2D(outputImage.ptr(), outputImage.step[0], d_output, pitch, width * channels * sizeof(unsigned char), height, hipMemcpyDeviceToHost);

    // �޸� ����
    hipFree(d_input);
    hipFree(d_output);
}

void callBilateralFilterCUDA(
    cv::Mat& inputImage,
    cv::Mat& outputImage,
    int kernelSize,
    float sigmaColor,
    float sigmaSpace
) {
    if (inputImage.empty()) {
        std::cerr << "�Է� �̹����� ��� �ֽ��ϴ�." << std::endl;
        return;
    }

    int width = inputImage.cols;
    int height = inputImage.rows;
    int channels = inputImage.channels();

    unsigned char* d_input;
    unsigned char* d_output;
    size_t pitch;

    // ��ġ �޸� �Ҵ� (pitch ���)
    hipMallocPitch(&d_input, &pitch, width * channels * sizeof(unsigned char), height);
    hipMallocPitch(&d_output, &pitch, width * channels * sizeof(unsigned char), height);

    // �Է� �̹����� ��ġ�� ����
    hipMemcpy2D(d_input, pitch, inputImage.ptr(), inputImage.step, width * channels * sizeof(unsigned char), height, hipMemcpyHostToDevice);

    // CUDA ��� �� �׸��� ũ�� ����
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Ŀ�� ����
    bilateralKernel << <gridSize, blockSize >> > (
        d_input, d_output, width, height, pitch, kernelSize, channels, sigmaColor, sigmaSpace
        );

    // ���� Ȯ��
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Ŀ�� ����: " << hipGetErrorString(err) << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        return;
    }

    // ��� �̹��� ����
    outputImage.create(height, width, inputImage.type());

    // ��ġ���� ȣ��Ʈ�� ��� �̹��� ����
    hipMemcpy2D(outputImage.ptr(), outputImage.step, d_output, pitch, width * channels * sizeof(unsigned char), height, hipMemcpyDeviceToHost);

    // ��ġ �޸� ����
    hipFree(d_input);
    hipFree(d_output);
}

void callSobelFilterCUDA(cv::Mat& inputImage, cv::Mat& outputImage) {
    int width = inputImage.cols;
    int height = inputImage.rows;
    int channels = inputImage.channels();

    size_t pitch;
    unsigned char* d_input;
    unsigned char* d_output;

    // Allocate CUDA memory
    hipMallocPitch(&d_input, &pitch, width * channels * sizeof(unsigned char), height);
    hipMallocPitch(&d_output, &pitch, width * channels * sizeof(unsigned char), height);

    // Copy input image to device memory
    hipMemcpy2D(d_input, pitch, inputImage.ptr(), inputImage.step[0], width * channels * sizeof(unsigned char), height, hipMemcpyHostToDevice);

    // Define CUDA block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    if (channels == 1 || channels == 3) {
        // Apply Sobel filter
        sobelFilterKernel << <gridSize, blockSize >> > (d_input, d_output, width, height, channels);
        hipDeviceSynchronize(); // Ensure kernel completion

        // Check for CUDA errors
        hipError_t cudaErr = hipGetLastError();
        if (cudaErr != hipSuccess) {
            std::cerr << "CUDA error: " << hipGetErrorString(cudaErr) << std::endl;
            hipFree(d_input);
            hipFree(d_output);
            return;
        }
    }
    else {
        std::cerr << "Unsupported number of channels: " << channels << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        return;
    }

    // Copy the result back to host memory
    outputImage.create(height, width, inputImage.type());
    hipMemcpy2D(outputImage.ptr(), outputImage.step[0], d_output, pitch, width * channels * sizeof(unsigned char), height, hipMemcpyDeviceToHost);

    // Free CUDA memory
    hipFree(d_input);
    hipFree(d_output);
}




void createGaussianKernel(float* kernel, int kernelSize, float sigma)
{
    int halfSize = kernelSize / 2;
    float sum = 0.0f;

    for (int i = -halfSize; i <= halfSize; ++i) {
        for (int j = -halfSize; j <= halfSize; ++j) {
            kernel[(i + halfSize) * kernelSize + (j + halfSize)] = expf(-(i * i + j * j) / (2.0f * sigma * sigma));
            sum += kernel[(i + halfSize) * kernelSize + (j + halfSize)];
        }
    }

    // Normalize the kernel
    for (int i = 0; i < kernelSize; ++i) {
        for (int j = 0; j < kernelSize; ++j) {
            kernel[i * kernelSize + j] /= sum;
        }
    }
}
