#include "hip/hip_runtime.h"
#include "imageProcessing.cuh"
#include <opencv2/core/cuda.hpp>

// ���� �Լ� ����
__device__ bool rotatePixel(int x, int y, int cols, int rows, int channels, const uchar* input, uchar* output) {
    if (x < cols && y < rows) {
        for (int c = 0; c < channels; ++c) {
            output[(x * rows + (rows - 1 - y)) * channels + c] = input[(y * cols + x) * channels + c];
        }
        return true;
    }
    return false;
}

// CUDA Ŀ��
__global__ void rotateImageCUDA(const uchar* input, uchar* output, int cols, int rows, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (rotatePixel(x, y, cols, rows, channels, input, output)) {
        // ó���� ������ ������ �߰�
    }
}


void callRotateImageCUDA(cv::Mat& inputImage) {
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    uchar* d_inputImage;
    uchar* d_outputImage;
    size_t imageSize = cols * rows * channels * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, imageSize);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        return;
    }

    err = hipMalloc(&d_outputImage, imageSize);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA malloc error: %s\n", hipGetErrorString(err));
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, imageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA memcpy error: %s\n", hipGetErrorString(err));
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    rotateImageCUDA << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows, channels);

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA kernel launch error: %s\n", hipGetErrorString(err));
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    err = hipMemcpy(inputImage.data, d_outputImage, imageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA memcpy error: %s\n", hipGetErrorString(err));
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}
