#include "hip/hip_runtime.h"
//imageProecssing.cu
#include "imageProcessing.cuh"

__device__ void rotatePixel(int x, int y, int cols, int rows, int channels, const unsigned char* input, unsigned char* output) {
    if (x < cols && y < rows) {
        for (int c = 0; c < channels; ++c) {
            output[(x * rows + (rows - 1 - y)) * channels + c] = input[(y * cols + x) * channels + c];
        }
    }
}

__device__ float gaussian(float x, float sigma) {
    return expf(-(x * x) / (2 * sigma * sigma));
}

__global__ void rotateImageKernel(const unsigned char* input, unsigned char* output, int cols, int rows, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        rotatePixel(x, y, cols, rows, channels, input, output);
    }
}

__global__ void resizeImageKernel(const unsigned char* input, unsigned char* output, int oldWidth, int oldHeight, int newWidth, int newHeight, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < newWidth && y < newHeight) {
        float x_ratio = oldWidth / (float)newWidth;
        float y_ratio = oldHeight / (float)newHeight;
        int px = floor(x * x_ratio);
        int py = floor(y * y_ratio);

        for (int c = 0; c < channels; ++c) {
            output[(y * newWidth + x) * channels + c] = input[(py * oldWidth + px) * channels + c];
        }
    }
}

__global__ void grayScaleImageKernel(const unsigned char* input, unsigned char* output, int cols, int rows) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int idx = y * cols + x;
        unsigned char b = input[idx * 3 + 0];
        unsigned char g = input[idx * 3 + 1];
        unsigned char r = input[idx * 3 + 2];
        output[idx] = 0.299f * r + 0.587f * g + 0.114f * b;  // �׷��̽����� ��ȯ
    }
}

__global__ void cannyEdgesKernel(const unsigned char* input, unsigned char* output, int cols, int rows) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int idx = y * cols + x;
        float gradientX = 0.0f, gradientY = 0.0f;

        // Calculate gradients (Sobel operators)
        if (x > 0 && x < cols - 1 && y > 0 && y < rows - 1) {
            gradientX = -1.0f * input[(y - 1) * cols + (x - 1)] + 1.0f * input[(y - 1) * cols + (x + 1)]
                - 2.0f * input[y * cols + (x - 1)] + 2.0f * input[y * cols + (x + 1)]
                - 1.0f * input[(y + 1) * cols + (x - 1)] + 1.0f * input[(y + 1) * cols + (x + 1)];

            gradientY = -1.0f * input[(y - 1) * cols + (x - 1)] - 2.0f * input[(y - 1) * cols + x] - 1.0f * input[(y - 1) * cols + (x + 1)]
                + 1.0f * input[(y + 1) * cols + (x - 1)] + 2.0f * input[(y + 1) * cols + x] + 1.0f * input[(y + 1) * cols + (x + 1)];
        }

        // Calculate gradient magnitude
        float gradientMagnitude = sqrtf(gradientX * gradientX + gradientY * gradientY);

        // Apply hysteresis thresholding to detect edges
        if (gradientMagnitude > 50) {  // Adjust this threshold as needed
            output[idx] = 255;
        }
        else {
            output[idx] = 0;
        }
    }
}

__global__ void gaussianBlurKernel(const unsigned char* input, unsigned char* output, int cols, int rows, int kernelSize, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int halfSize = kernelSize / 2;
        float sum = 0.0f;

        // Apply Gaussian blur using the kernel size
        for (int c = 0; c < channels; ++c) {
            sum = 0.0f;

            for (int i = -halfSize; i <= halfSize; ++i) {
                for (int j = -halfSize; j <= halfSize; ++j) {
                    int offsetX = x + i;
                    int offsetY = y + j;

                    if (offsetX >= 0 && offsetX < cols && offsetY >= 0 && offsetY < rows) {
                        float weight = exp(-(i * i + j * j) / (2.0f * kernelSize * kernelSize));
                        sum += weight * input[(offsetY * cols + offsetX) * channels + c];
                    }
                }
            }

            output[(y * cols + x) * channels + c] = static_cast<unsigned char>(sum);
        }
    }
}

__device__ void device_sort(unsigned char* values, int length) {
    // Sorting implementation using device-specific method
    // Example: bubble sort
    for (int i = 0; i < length - 1; ++i) {
        for (int j = 0; j < length - i - 1; ++j) {
            if (values[j] > values[j + 1]) {
                unsigned char temp = values[j];
                values[j] = values[j + 1];
                values[j + 1] = temp;
            }
        }
    }
}

__global__ void medianFilterKernel(const unsigned char* input, unsigned char* output, int cols, int rows, int channels, int kernelSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        int halfSize = kernelSize / 2;
        int kernelLength = kernelSize * kernelSize;

        // ���� ������ �ȼ��� �ε��� ���
        int idx = (y * cols + x) * channels;

        // �߰����� ã�� ���� �ӽ� ���� ����
        unsigned char values[25]; // �ִ� kernelSize�� 5�� ����

        // �� ä�ο� ���� median ���� ����
        for (int c = 0; c < channels; ++c) {
            // ���� �ʱ�ȭ
            for (int i = 0; i < kernelLength; ++i) {
                int offsetX = x + (i % kernelSize) - halfSize;
                int offsetY = y + (i / kernelSize) - halfSize;

                // ��� ó��
                offsetX = max(0, min(cols - 1, offsetX));
                offsetY = max(0, min(rows - 1, offsetY));

                values[i] = input[(offsetY * cols + offsetX) * channels + c];
            }

            // ���� ���� �� �߰��� ��� (���� ���� �˰��� ȣ��)
            device_sort(values, kernelLength);

            output[idx + c] = values[kernelLength / 2];
        }
    }
}

__global__ void laplacianFilterKernel(const unsigned char* input, unsigned char* output,
    int cols, int rows, size_t pitch, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        // Laplacian ���� ���
        int sum = 0;
        sum += input[y * pitch + x * channels]; // ���� �ȼ�

        if (x > 0)
            sum += input[y * pitch + (x - 1) * channels]; // ���� �ȼ�

        if (x < cols - 1)
            sum += input[y * pitch + (x + 1) * channels]; // ������ �ȼ�

        if (y > 0)
            sum += input[(y - 1) * pitch + x * channels]; // ���� �ȼ�

        if (y < rows - 1)
            sum += input[(y + 1) * pitch + x * channels]; // �Ʒ��� �ȼ�

        output[y * pitch + x * channels] = static_cast<unsigned char>(sum / 5); // Laplacian ���� ���
    }
}

__global__ void bilateralKernel(const unsigned char* input, unsigned char* output, int width, int height, int kernelSize, int channels, float sigmaColor, float sigmaSpace) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int half = kernelSize / 2;
    float colorCoeff = -0.5f / (sigmaColor * sigmaColor);
    float spaceCoeff = -0.5f / (sigmaSpace * sigmaSpace);

    for (int c = 0; c < channels; ++c) {
        float sum = 0;
        float norm = 0;

        for (int i = -half; i <= half; ++i) {
            for (int j = -half; j <= half; ++j) {
                int neighborX = min(max(x + j, 0), width - 1);
                int neighborY = min(max(y + i, 0), height - 1);

                int idx = (y * width + x) * channels + c;
                int nIdx = (neighborY * width + neighborX) * channels + c;

                float spaceDist = (i * i + j * j) * spaceCoeff;
                float colorDist = (input[idx] - input[nIdx]) * (input[idx] - input[nIdx]) * colorCoeff;

                float weight = expf(spaceDist + colorDist);
                sum += weight * input[nIdx];
                norm += weight;
            }
        }
        output[(y * width + x) * channels + c] = min(max(int(sum / norm), 0), 255);
    }
}

__global__ void sobelFilterKernel(const unsigned char* input, unsigned char* output,
    int cols, int rows, int channels) {
    // �����尡 ó���� �̹����� �ȼ� ��ġ ���
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < cols && y < rows) {
        // ���� �� ���� ������ �Һ� ����ũ ����
        const int sobelX[3][3] = { {-1, 0, 1},
                                   {-2, 0, 2},
                                   {-1, 0, 1} };

        const int sobelY[3][3] = { {-1, -2, -1},
                                   {0, 0, 0},
                                   {1, 2, 1} };

        float gradX = 0.0f;
        float gradY = 0.0f;

        // �� ä�ο� ���� �Һ� ���� ���
        for (int c = 0; c < channels; ++c) {
            for (int i = -1; i <= 1; ++i) {
                for (int j = -1; j <= 1; ++j) {
                    int offsetX = x + j;
                    int offsetY = y + i;

                    if (offsetX >= 0 && offsetX < cols && offsetY >= 0 && offsetY < rows) {
                        int pixelIndex = (offsetY * cols + offsetX) * channels + c;
                        gradX += sobelX[i + 1][j + 1] * input[pixelIndex];
                        gradY += sobelY[i + 1][j + 1] * input[pixelIndex];
                    }
                }
            }
        }

        // �׷����Ʈ ũ�� ��� (���״�Ʃ��)
        float magnitude = sqrtf(gradX * gradX + gradY * gradY);

        // ���� �׷����Ʈ �� (0-255 ������ Ŭ����)
        for (int c = 0; c < channels; ++c) {
            output[(y * cols + x) * channels + c] = static_cast<unsigned char>(min(magnitude, 255.0f));
        }
    }
}


void callRotateImageCUDA(cv::Mat& inputImage, cv::Mat& outputImage) {
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t imageSize = cols * rows * channels * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, imageSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, imageSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, imageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    rotateImageKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows, channels);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    outputImage.create(rows, cols, inputImage.type());

    err = hipMemcpy(outputImage.data, d_outputImage, imageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

void callZoomImageCUDA(cv::Mat& inputImage, cv::Mat& outputImage, int newWidth, int newHeight) {
    int oldWidth = inputImage.cols;
    int oldHeight = inputImage.rows;
    int channels = inputImage.channels();

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t oldImageSize = oldWidth * oldHeight * channels * sizeof(uchar);
    size_t newImageSize = newWidth * newHeight * channels * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, oldImageSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, newImageSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, oldImageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((newWidth + threadsPerBlock.x - 1) / threadsPerBlock.x, (newHeight + threadsPerBlock.y - 1) / threadsPerBlock.y);

    resizeImageKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, oldWidth, oldHeight, newWidth, newHeight, channels);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    outputImage.create(newHeight, newWidth, inputImage.type());

    err = hipMemcpy(outputImage.data, d_outputImage, newImageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

void callGrayScaleImageCUDA(cv::Mat& inputImage, cv::Mat& outputImage) {
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    if (channels != 3) {
        std::cerr << "Input image must be a 3-channel BGR image." << std::endl;
        return;
    }

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t inputSize = cols * rows * channels * sizeof(uchar);
    size_t outputSize = cols * rows * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, inputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, inputSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    grayScaleImageKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    outputImage.create(rows, cols, CV_8UC1);
    err = hipMemcpy(outputImage.data, d_outputImage, outputSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

void callCannyEdgesCUDA(cv::Mat& inputImage, cv::Mat& outputImage) {
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    //if (channels != 3) {
    //    std::cerr << "Input image must be a 3-channel BGR image." << std::endl;
    //    return;
    //}

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t inputSize = cols * rows * channels * sizeof(uchar);
    size_t outputSize = cols * rows * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, inputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, inputSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    cannyEdgesKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    outputImage.create(rows, cols, CV_8UC1);
    err = hipMemcpy(outputImage.data, d_outputImage, outputSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

void callGaussianBlurCUDA(cv::Mat& inputImage, cv::Mat& outputImage, int kernelSize) {
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t inputSize = cols * rows * channels * sizeof(uchar);
    size_t outputSize = cols * rows * channels * sizeof(uchar);

    hipError_t err;
    err = hipMalloc(&d_inputImage, inputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    err = hipMemcpy(d_inputImage, inputImage.data, inputSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    gaussianBlurKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows, kernelSize, channels);

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    hipDeviceSynchronize();

    outputImage.create(rows, cols, inputImage.type());
    err = hipMemcpy(outputImage.data, d_outputImage, outputSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
    }

    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

void callMedianFilterCUDA(cv::Mat & inputImage, cv::Mat& outputImage)
{
    // �̹����� �ʺ�, ����, ä�� �� Ȯ��
    int cols = inputImage.cols;
    int rows = inputImage.rows;
    int channels = inputImage.channels();

    // GPU �޸� �Ҵ�
    uchar* d_inputImage = nullptr;
    uchar* d_outputImage = nullptr;
    size_t inputSize = cols * rows * channels * sizeof(uchar);
    size_t outputSize = cols * rows * channels * sizeof(uchar);

    hipError_t err;

    err = hipMalloc(&d_inputImage, inputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_outputImage, outputSize);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        return;
    }

    // CPU���� GPU�� �̹��� ������ ����
    err = hipMemcpy(d_inputImage, inputImage.data, inputSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy H2D error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    // CUDA ������ ���� ����
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // ä�� ���� ���� ������ Ŀ�� ����
    if (channels == 1 || channels == 3) {
        medianFilterKernel << <numBlocks, threadsPerBlock >> > (d_inputImage, d_outputImage, cols, rows, channels, 5);
    }
    else {
        std::cerr << "Unsupported number of channels: " << channels << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    // CUDA Ŀ�� ���� ���� Ȯ��
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_inputImage);
        hipFree(d_outputImage);
        return;
    }

    // GPU���� ó�� �Ϸ� ���
    hipDeviceSynchronize();

    // GPU���� CPU�� ��� �̹��� ����
    cv::Mat gpuOutputImage(rows, cols, inputImage.type());
    err = hipMemcpy(gpuOutputImage.data, d_outputImage, outputSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy D2H error: " << hipGetErrorString(err) << std::endl;
    }
    //else {
    //    inputImage = outputImage.clone();
    //}
    outputImage = gpuOutputImage.clone();

    // �޸� ����
    hipFree(d_inputImage);
    hipFree(d_outputImage);
}

void callLaplacianFilterCUDA(cv::Mat& inputImage) {
    int width = inputImage.cols;
    int height = inputImage.rows;
    int channels = inputImage.channels();

    unsigned char* d_input;
    unsigned char* d_output;
    size_t pitch;

    hipMallocPitch(&d_input, &pitch, width * channels * sizeof(unsigned char), height);
    hipMallocPitch(&d_output, &pitch, width * channels * sizeof(unsigned char), height);

    hipMemcpy2D(d_input, pitch, inputImage.ptr(), width * channels * sizeof(unsigned char), width * channels * sizeof(unsigned char), height, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    laplacianFilterKernel << <gridSize, blockSize >> > (d_input, d_output, width, height, pitch, channels);

    hipMemcpy2D(inputImage.ptr(), width * channels * sizeof(unsigned char), d_output, pitch, width * channels * sizeof(unsigned char), height, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

void callBilateralFilterCUDA(cv::Mat& inputImage, int kernelSize, float sigmaColor, float sigmaSpace) {
    int width = inputImage.cols;
    int height = inputImage.rows;
    int channels = inputImage.channels();

    cv::Mat outputImage(height, width, inputImage.type());

    unsigned char* d_input;
    unsigned char* d_output;
    size_t pitch;

    hipMallocPitch(&d_input, &pitch, width * channels * sizeof(unsigned char), height);
    hipMallocPitch(&d_output, &pitch, width * channels * sizeof(unsigned char), height);

    hipMemcpy2D(d_input, pitch, inputImage.ptr(), width * channels * sizeof(unsigned char), width * channels * sizeof(unsigned char), height, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    bilateralKernel << <gridSize, blockSize >> > (d_input, d_output, width, height, kernelSize, channels, sigmaColor, sigmaSpace);

    hipMemcpy2D(inputImage.ptr(), width * channels * sizeof(unsigned char), d_output, pitch, width * channels * sizeof(unsigned char), height, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

void callSobelFilterCUDA(cv::Mat& inputImage) {
    // �Է� �̹����� �ʺ�, ����, ä�� ��
    int width = inputImage.cols;
    int height = inputImage.rows;
    int channels = inputImage.channels();

    // CUDA �޸� �Ҵ� �� ����
    unsigned char* d_input, * d_output;
    size_t pitch;
    hipMallocPitch(&d_input, &pitch, width * channels * sizeof(unsigned char), height);
    hipMallocPitch(&d_output, &pitch, width * channels * sizeof(unsigned char), height);

    hipMemcpy2D(d_input, pitch, inputImage.ptr(), width * channels * sizeof(unsigned char), width * channels * sizeof(unsigned char), height, hipMemcpyHostToDevice);

    // CUDA ��� �� �׸��� ����
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // CUDA Ŀ�� ȣ��
    sobelFilterKernel << <gridSize, blockSize >> > (d_input, d_output, width, height, channels);

    // CUDA���� ó���� ����� ȣ��Ʈ�� ����
    hipMemcpy2D(inputImage.ptr(), width * channels * sizeof(unsigned char), d_output, pitch, width * channels * sizeof(unsigned char), height, hipMemcpyDeviceToHost);

    // �޸� ����
    hipFree(d_input);
    hipFree(d_output);
}